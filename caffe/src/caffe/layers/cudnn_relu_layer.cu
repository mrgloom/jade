
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.relu_param().negative_slope() != 0) {
    return ReLULayer<Dtype>::Forward_gpu(bottom, top);
  }

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  Dtype alpha = 1.0;
  Dtype beta  = 0.0;

  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        CUDNN_ACTIVATION_RELU,
        reinterpret_cast<void *>(&alpha),
        this->bottom_desc_, bottom_data,
        reinterpret_cast<void *>(&beta),
        this->top_desc_, top_data));
}

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.relu_param().negative_slope() != 0) {
    return ReLULayer<Dtype>::Backward_gpu(top, propagate_down, bottom);
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  Dtype alpha = 1.0;
  Dtype beta = 0.0;

  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        CUDNN_ACTIVATION_RELU,
        reinterpret_cast<void *>(&alpha),
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        reinterpret_cast<void *>(&beta),
        this->bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNReLULayer);

}  // namespace caffe
#endif
